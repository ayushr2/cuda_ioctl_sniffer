#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <signal.h>
#include <unistd.h>
#include <assert.h>
#include <sys/mman.h>

//#define BROKEN

extern "C" {
extern const unsigned long long fatbinData[351];
}

__global__
void saxpy(int n, float a, float *x, float *y, int bob)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void dump_gpu_ctrl() {
  printf("***** read\n");
  uint32_t *ptr = (uint32_t*)0x200400000;
  while (ptr != (uint32_t*)0x203600000) { if (*ptr != 0) printf("%p: %16lx\n", ptr, *ptr); ++ptr; }
}

int main(int argc, char *argv[]) {
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  printf("***** init\n");
  hipInit(0);

  printf("***** get device\n");
  hipDevice_t pdev;
  hipDeviceGet(&pdev, 0);

  printf("***** create context\n");
  hipCtx_t pctx;
  hipCtxCreate(&pctx, 0, pdev);

  printf("***** get function\n");
  hipModule_t mod = 0;
  hipModuleLoadFatBinary(&mod, fatbinData);

  // ihipModuleSymbol_t*
  hipFunction_t saxpy_f = 0;
  hipModuleGetFunction(&saxpy_f, mod, "_Z5saxpyifPfS_i");
  printf("function %p\n", saxpy_f);
  assert(saxpy_f != 0);

  printf("***** print memory\n");
  size_t free_byte, total_byte;
  hipMemGetInfo(&free_byte, &total_byte);
  printf("%.2f MB used\n", (total_byte-free_byte)/1e6);

  printf("***** entry malloc\n");
  hipMalloc(&d_x, N*sizeof(float)); 
  printf("***** entry malloc 2\n");
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  printf("***** entry memcpy\n");
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  printf("***** unmap\n");
  //raise(SIGTRAP);
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  int bob = 0x1337f00d;
  float ratio = 2.0f;
  void *args[] = { &N, &ratio, &d_x, &d_y, &bob};

  munmap((void*)0x200200000, 0x200400000-0x200200000);    // /dev/nvidiactl

#ifdef BROKEN
  // these are needed for launch
  // * Nvidia device node(nvidia#) maps device's BAR memory,
  // * Nvidia control node(nvidiactrl) maps system memory.
  munmap((void*)0x200400000, 0x200600000-0x200400000);    // /dev/nvidia0    NEEDED
  munmap((void*)0x200600000, 0x203600000-0x200600000);    // /dev/nvidiactl  NEEDED
  //munmap((void*)0x204600000, 0x204800000-0x204600000);    // /dev/nvidiactl  NEEDED (READ ONLY)
  mprotect((void*)0x204600000, 0x204800000-0x204600000, PROT_READ);

  void *ret = mmap((void*)0x200400000, 0x203600000-0x200400000, PROT_READ | PROT_WRITE, MAP_FIXED | MAP_SHARED | MAP_ANON, -1, 0);
  assert(ret == (void*)0x200400000);
#endif
  memset((void*)0x200400000, 0, 0x203600000-0x200400000);

  munmap((void*)0x204800000, 0x204a00000-0x204800000);    // /dev/nvidiactl
  munmap((void*)0x204a00000, 0x204c00000-0x204a00000);    // /dev/nvidia-uvm
  munmap((void*)0x204c00000, 0x204e00000-0x204c00000);    // /dev/nvidiactl
  //munmap((void*)0x205000000, 0x205200000-0x205000000);    // /dev/nvidiactl  NEEDED AFTER LAUNCH
  mprotect((void*)0x205000000, 0x205200000-0x205000000, PROT_READ);

  /*char buf[0x10000];
  FILE *f = fopen("/proc/self/maps", "rb");
  //FILE *f = fopen("/proc/self/pagemap", "rb");
  buf[fread(buf, 1, sizeof(buf), f)] = '\0';
  printf("%s\n", buf);*/

  //while (1) sleep(1);

  // calls into /lib/x86_64-linux-gnu/libcuda.so.515.43.04
  printf("***** launch\n");
  hipModuleLaunchKernel(saxpy_f, (N+255)/256, 1, 1, 256, 1, 1, 0, 0, args, NULL);
  hipStreamSynchronize(0);
  dump_gpu_ctrl();

  //printf("***** sync\n");
  //memset((void*)0x200400000, 0, 0x203600000-0x200400000);
  //hipStreamSynchronize(0);
  //dump_gpu_ctrl();

  printf("***** exit memcpy %p -> %p\n", d_y, y);
  memset((void*)0x200400000, 0, 0x203600000-0x200400000);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  dump_gpu_ctrl();

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  if (maxError > 0.01) { printf("FAILLLLLLLLED\n"); exit(-1); }

  printf("***** print memory 2\n");
  hipMemGetInfo(&free_byte, &total_byte);
  printf("%.2f MB used\n", (total_byte-free_byte)/1e6);

  printf("***** exit free\n");
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

}
