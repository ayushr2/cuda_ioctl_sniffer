#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <signal.h>
#include <unistd.h>
#include <assert.h>
#include <sys/mman.h>
#include "helpers.h"

//#define BROKEN
//#define DUMP_MAPS
extern "C" {
extern const unsigned long long fatbinData[351];
//extern const unsigned long long fatbinData[372];
}

__global__
void saxpy(int n, float a, float *x, float *y, int bob) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char *argv[]) {
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  printf("***** init\n");
  hipInit(0);

  printf("***** get device\n");
  hipDevice_t pdev;
  hipDeviceGet(&pdev, 0);

  printf("***** create context\n");
  hipCtx_t pctx;
  hipCtxCreate(&pctx, 0, pdev);

  printf("***** get function\n");
  hipModule_t mod = 0;

  hipModuleLoadFatBinary(&mod, fatbinData);
  /*dump_gpu_ctrl();
  //dump_command_buffer(0x200400000);
  dump_command_buffer(0x200400008);
  exit(0);*/

  hipFunction_t saxpy_f = 0;
  hipModuleGetFunction(&saxpy_f, mod, "_Z5saxpyifPfS_i");
  printf("function %p\n", saxpy_f);
  assert(saxpy_f != 0);

  printf("***** entry malloc\n");
  //hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc((hipDeviceptr_t*)&d_x, N*sizeof(float)); 
  printf("***** entry malloc 2\n");
  hipMalloc((hipDeviceptr_t*)&d_y, N*sizeof(float));
  printf("%p %p\n", d_x, d_y);

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  printf("***** entry memcpy 1\n");
  //clear_gpu_ctrl();
  cuMemcpy((hipDeviceptr_t)d_x, (hipDeviceptr_t)x, N*sizeof(float));

  //hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  /*dump_gpu_ctrl();
  dump_command_buffer(0x200418158);*/

  printf("***** entry memcpy 2\n");
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  printf("***** unmap\n");
  //raise(SIGTRAP);
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  int bob = 0x1337f00d;
  float ratio = 2.0f;
  void *args[] = { &N, &ratio, &d_x, &d_y, &bob};

  munmap((void*)0x200200000, 0x200400000-0x200200000);    // /dev/nvidiactl

#ifdef BROKEN
  // these are needed for launch
  // * Nvidia device node(nvidia#) maps device's BAR memory,
  // * Nvidia control node(nvidiactrl) maps system memory.
  munmap((void*)0x200400000, 0x200600000-0x200400000);    // /dev/nvidia0    NEEDED
  munmap((void*)0x200600000, 0x203600000-0x200600000);    // /dev/nvidiactl  NEEDED
  //munmap((void*)0x204600000, 0x204800000-0x204600000);    // /dev/nvidiactl  NEEDED (READ ONLY)
  mprotect((void*)0x204600000, 0x204800000-0x204600000, PROT_READ);

  void *ret = mmap((void*)0x200400000, 0x203600000-0x200400000, PROT_READ | PROT_WRITE, MAP_FIXED | MAP_SHARED | MAP_ANON, -1, 0);
  assert(ret == (void*)0x200400000);
#endif
  //memset((void*)0x200400000, 0, 0x203600000-0x200400000);
  mprotect((void*)0x204600000, 0x204800000-0x204600000, PROT_READ);

  munmap((void*)0x204800000, 0x204a00000-0x204800000);    // /dev/nvidiactl
  munmap((void*)0x204a00000, 0x204c00000-0x204a00000);    // /dev/nvidia-uvm
  munmap((void*)0x204c00000, 0x204e00000-0x204c00000);    // /dev/nvidiactl
  //munmap((void*)0x205000000, 0x205200000-0x205000000);    // /dev/nvidiactl  NEEDED AFTER LAUNCH
  mprotect((void*)0x205000000, 0x205200000-0x205000000, PROT_READ);

#ifdef DUMP_MAPS
  dump_proc_self_maps();
#endif

  //while (1) sleep(1);

  // calls into /lib/x86_64-linux-gnu/libcuda.so.515.43.04
  printf("***** launch program\n");
  hipModuleLaunchKernel(saxpy_f, (N+255)/256, 1, 1, 256, 1, 1, 0, 0, args, NULL);
  hipStreamSynchronize(0);
  //sleep(1);
  //dump_gpu_ctrl();
  //dump_command_buffer(0x200400418);

  // 65453c
  // 35602
  /**((uint64_t*)0x200400418) = 0x65453c;
  *((uint64_t*)0x20040041C) = 0x35602;*/


  //uint32_t *ep = (uint32_t *)(*((uint64_t*)0x200402040) & 0xFFFFFFFFFF);
  //printf("dumping %p -> %p\n", sp, ep);

  /*while (sp != ep) {
    printf("0x%X,", *sp);
    sp++;
  }
  printf("\n");*/

  //printf("***** sync\n");
  //memset((void*)0x200400000, 0, 0x203600000-0x200400000);
  //hipStreamSynchronize(0);
  //dump_gpu_ctrl();

  printf("***** exit memcpy %p -> %p\n", d_y, y);
  //memset((void*)0x200400000, 0, 0x203600000-0x200400000);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  /*dump_gpu_ctrl();
  dump_command_buffer(*((uint64_t*)0x200424008));
  dump_command_buffer(*((uint64_t*)0x200424010));
  dump_command_buffer(*((uint64_t*)0x200424018));
  dump_command_buffer(*((uint64_t*)0x200424020));*/
  
  //) & 0xFFFFFFFFFF));
  //dump_command_buffer((uint32_t *)(*((uint64_t*)0x200424008) & 0xFFFFFFFFFF));

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  if (maxError > 0.01) { printf("FAILLLLLLLLED\n"); exit(-1); }

  /*printf("***** dump progrem\n");
  char tmp[0x100] = {0};
  int ret = cuMemcpy((hipDeviceptr_t)tmp, (hipDeviceptr_t)0x7FFFE6FB7900, 0x100);
  printf("copy %d\n", ret);
  hexdump(tmp, 0x100);*/

  printf("***** exit free\n");
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

