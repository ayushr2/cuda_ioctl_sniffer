#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <signal.h>

extern "C" {
extern const unsigned long long fatbinData[346];
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


int main(int argc, char *argv[]) {
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  printf("***** init\n");
  hipInit(0);

  printf("***** get device\n");
  hipDevice_t pdev;
  hipDeviceGet(&pdev, 0);

  printf("***** create context\n");
  hipCtx_t pctx;
  hipCtxCreate(&pctx, 0, pdev);

  printf("***** get function\n");
  hipModule_t mod = 0;
  hipModuleLoadFatBinary(&mod, fatbinData);

  hipFunction_t saxpy_f = 0;
  hipModuleGetFunction(&saxpy_f, mod, "_Z5saxpyifPfS_");
  printf("function 0x%X\n", saxpy_f);

  printf("***** print memory\n");
  size_t free_byte, total_byte;
  hipMemGetInfo(&free_byte, &total_byte);
  printf("%.2f MB used\n", (total_byte-free_byte)/1e6);

  printf("***** entry malloc\n");
  hipMalloc(&d_x, N*sizeof(float)); 
  printf("***** entry malloc 2\n");
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  printf("***** entry memcpy\n");
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  printf("***** launch\n");
  //raise(SIGTRAP);
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  float ratio = 2.0f;
  void *args[] = { &N, &ratio, &d_x, &d_y };
  hipModuleLaunchKernel(saxpy_f, (N+255)/256, 1, 1, 256, 1, 1, 0, 0, args, NULL);

  printf("***** exit memcpy\n");
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  printf("***** print memory 2\n");
  hipMemGetInfo(&free_byte, &total_byte);
  printf("%.2f MB used\n", (total_byte-free_byte)/1e6);

  printf("***** exit free\n");
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

}
