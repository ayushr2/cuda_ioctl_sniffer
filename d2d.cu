#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
// dmesg -w | grep -Ei "p2p|fault"

int main(int argc, char *argv[]) {
  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device %d name: %s\n", i, prop.name);
  }

  int g0=0, g1=1;
  //int g0=1, g1=0;

  printf("***** malloc %d\n", g1);
  hipSetDevice(g1);
  float *b = NULL;
  hipMalloc(&b, 0x13370);

  printf("***** malloc %d\n", g0);
  hipSetDevice(g0);
  float *a = NULL;
  hipMalloc(&a, 0x13370);
  hipSetDevice(g1);

  printf("***** enable p2p\n");
  hipError_t err = hipDeviceEnablePeerAccess(g0, 0);
  assert(err == hipSuccess);

  printf("***** hipMemcpyDtoD %p -> %p\n", a, b);
  hipMemcpyDtoD((hipDeviceptr_t)b, (hipDeviceptr_t)a, 0x1000);

  printf("***** hipMemcpyDtoD %p -> %p\n", b, a);
  hipMemcpyDtoD((hipDeviceptr_t)a, (hipDeviceptr_t)b, 0x1000);

  printf("***** done\n");
}
