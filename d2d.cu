#include <stdio.h>
#include <hip/hip_runtime.h>
// dmesg -w | grep -Ei "p2p|fault"

int main(int argc, char *argv[]) {
  printf("***** malloc 0\n");
  hipSetDevice(0);
  float *a = NULL;
  hipMalloc(&a, 0x10000);

  printf("***** malloc 1\n");
  hipSetDevice(1);
  float *b = NULL;
  hipMalloc(&b, 0x10000);

  printf("***** enable p2p\n");
  hipDeviceEnablePeerAccess(0, 0);

  printf("***** cuMemcpyDtoD\n");
  hipMemcpyDtoD((hipDeviceptr_t)a, (hipDeviceptr_t)b, 0x1);

  printf("***** done\n");
}